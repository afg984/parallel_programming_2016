#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>
#include <chrono>
#include <fstream>

#define XB 24
const int INF = 10000000;

int n, m;  // Number of vertices, edges
int R;
size_t devPitch, hostPitch;
unsigned* hostPtr;

#define check(err) __check(err, __LINE__)
void __check(hipError_t err, int line) {
    if (err) {
        fprintf(stderr, "%d:%s\n", line, hipGetErrorString(err));
        abort();
    }
}

#define dist(i, j) (hostPtr[(i)*n + (j)])

void sincelast(const char* message = 0) {
    static auto last = std::chrono::high_resolution_clock::now();
    auto now = std::chrono::high_resolution_clock::now();
    fprintf(stderr, "[%d] %16s  %lf\n", R, message,
            ((std::chrono::duration<double>)(now - last)).count());
    last = now;
}

void input(char* inFileName) {
    FILE* infile = fopen(inFileName, "r");
    fscanf(infile, "%d %d", &n, &m);

    hostPitch = sizeof(int) * n;
    check(hipHostMalloc(&hostPtr, hostPitch * n));

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j)
                dist(i, j) = 0;
            else
                dist(i, j) = INF;
        }
    }

    while (--m >= 0) {
        int a, b, v;
        fscanf(infile, "%u%u%u", &a, &b, &v);
        dist(a - 1, b - 1) = v;
    }
}

void output(char* outFileName) {
    std::ofstream fout(outFileName);
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (dist(i, j) >= INF)
                fout << "INF ";
            else
                fout << dist(i, j) << " ";
        }
        fout << "\n";
    }
}


#define safeat(i, j) ((i) < n and (j) < n ? at(i, j): INF)
#define at(i, j) (((int*)((char*)devPtr + (i)*devPitch))[j])
#define mifn(p, q)                \
    {                             \
        if ((p) > (q)) (p) = (q); \
    }

__global__ void phase1(unsigned* devPtr, int r, int n, int kmin, int devPitch) {
    int i = kmin + threadIdx.y;
    int j = kmin + threadIdx.x;
    if (i >= n or j >= n) {
        return;
    }
    int kmax = kmin + blockDim.x < n ? blockDim.x : n - kmin;
    __shared__ unsigned pivot[XB][XB];
    pivot[threadIdx.y][threadIdx.x] = at(i, j);
    for (int k = 0; k < kmax; ++k) {
        __syncthreads();
        mifn(pivot[threadIdx.y][threadIdx.x],
             pivot[threadIdx.y][k] + pivot[k][threadIdx.x]);
    }
    at(i, j) = pivot[threadIdx.y][threadIdx.x];
}

__global__ void phase2a(unsigned* devPtr, int r, int n, int kmin,
                        int devPitch) {
    if (blockIdx.y == r) {
        return;
    }
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = r * blockDim.x + threadIdx.x;
    int kmax = kmin + blockDim.x < n ? blockDim.x : n - kmin;
    __shared__ unsigned pivot[XB][XB], block[XB][XB];
    pivot[threadIdx.y][threadIdx.x] =
        safeat(kmin + threadIdx.y, kmin + threadIdx.x);
    block[threadIdx.y][threadIdx.x] = safeat(i, j);
    if (i >= n or j >= n) {
        return;
    }
    for (int k = 0; k < kmax; ++k) {
        __syncthreads();
        mifn(block[threadIdx.y][threadIdx.x],
             block[threadIdx.y][k] + pivot[k][threadIdx.x]);
    }
    at(i, j) = block[threadIdx.y][threadIdx.x];
}

__global__ void phase2b(unsigned* devPtr, int r, int n, int kmin,
                        int devPitch) {
    if (blockIdx.x == r) {
        return;
    }
    int i = r * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int kmax = kmin + blockDim.x < n ? blockDim.x : n - kmin;
    __shared__ unsigned pivot[XB][XB], block[XB][XB];
    pivot[threadIdx.y][threadIdx.x] =
        safeat(kmin + threadIdx.y, kmin + threadIdx.x);
    block[threadIdx.y][threadIdx.x] = safeat(i, j);
    if (i >= n or j >= n) {
        return;
    }
    for (int k = 0; k < kmax; ++k) {
        __syncthreads();
        mifn(block[threadIdx.y][threadIdx.x],
             block[k][threadIdx.x] + pivot[threadIdx.y][k]);
    }
    at(i, j) = block[threadIdx.y][threadIdx.x];
}

__global__ void phase3(unsigned* devPtr, int r, int n, int kmin, int devPitch, int ioff) {
    if (blockIdx.x == r or blockIdx.y + ioff == r) {
        return;
    }
    int i = (blockIdx.y + ioff) * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int kmax = kmin + blockDim.x < n ? blockDim.x : n - kmin;
    __shared__ unsigned iref[XB][XB], jref[XB][XB];
    iref[threadIdx.y][threadIdx.x] = safeat(i, kmin + threadIdx.x);
    jref[threadIdx.y][threadIdx.x] = safeat(kmin + threadIdx.y, j);
    if (i >= n or j >= n) {
        return;
    }
    int local = at(i, j);
    __syncthreads();
    for (int k = 0; k < kmax; ++k) {
        mifn(local, iref[threadIdx.y][k] + jref[k][threadIdx.x]);
    }
    at(i, j) = local;
}

#undef at

int ceil(int a, int b) { return (a + b - 1) / b; }

void blockfw(int B) {
    unsigned* devPtr;
    check(hipMallocPitch(&devPtr, &devPitch, hostPitch, n));
    check(hipMemcpy2D(devPtr, devPitch, hostPtr, hostPitch,
                       hostPitch, n, hipMemcpyHostToDevice));
    sincelast("hostToDevice");
    int rounds = ceil(n, B);
    int rdown = rounds / 2;
    int rup = (rounds + 1) / 2;
    for (int r = 0; r < rounds; ++r) {
        int kmin = r * B;
        if (R == 0) {
            phase1<<<1, dim3(B, B)>>>(devPtr, r, n, kmin, devPitch);
            phase2a<<<dim3(1, rounds), dim3(B, B)>>>(devPtr, r, n, kmin,
                                                     devPitch);
            phase2b<<<dim3(rounds, 1), dim3(B, B)>>>(devPtr, r, n, kmin,
                                                     devPitch);
            phase3<<<dim3(rounds, rup), dim3(B, B)>>>(devPtr, r, n, kmin,
                                                      devPitch, rdown);
            if (rdown) {
                MPI_Request req;
                MPI_Irecv(hostPtr, hostPitch * rdown * B, MPI_CHAR, 1, 0, MPI_COMM_WORLD, &req);
                check(hipMemcpy2D(
                    (char*)hostPtr + hostPitch * rdown * B, hostPitch,
                    (char*)devPtr + devPitch * rdown * B, devPitch,
                    hostPitch, n - rdown * B, hipMemcpyDeviceToHost));
                MPI_Send((char*)hostPtr + hostPitch * rdown * B, hostPitch * (n - rdown * B), MPI_CHAR, 1, 0, MPI_COMM_WORLD);
                MPI_Wait(&req, MPI_STATUS_IGNORE);
                check(hipMemcpy2D(devPtr, devPitch, hostPtr, hostPitch,
                                   hostPitch, rdown * B, hipMemcpyHostToDevice));
            }
        } else if (rdown and R == 1) {
            phase1<<<1, dim3(B, B)>>>(devPtr, r, n, kmin, devPitch);
            phase2a<<<dim3(1, rounds), dim3(B, B)>>>(devPtr, r, n, kmin,
                                                     devPitch);
            phase2b<<<dim3(rounds, 1), dim3(B, B)>>>(devPtr, r, n, kmin,
                                                     devPitch);
            phase3<<<dim3(rounds, rdown), dim3(B, B)>>>(devPtr, r, n, kmin,
                                                        devPitch, 0);

            MPI_Request req;
            MPI_Irecv((char*)hostPtr + hostPitch * rdown * B, hostPitch * (n - rdown * B), MPI_CHAR, 0, 0, MPI_COMM_WORLD, &req);
            check(hipMemcpy2D(hostPtr, hostPitch, devPtr, devPitch,
                               hostPitch, rdown * B, hipMemcpyDeviceToHost));
            MPI_Send(hostPtr, hostPitch * rdown * B, MPI_CHAR, 0, 0, MPI_COMM_WORLD);
            MPI_Wait(&req, MPI_STATUS_IGNORE);
            check(hipMemcpy2D(
                (char*)devPtr + devPitch * rdown * B, devPitch,
                (char*)hostPtr + hostPitch * rdown * B, hostPitch, hostPitch,
                n - rdown * B, hipMemcpyHostToDevice));
        }
    }
    check(hipDeviceSynchronize());
    sincelast("compute");
    check(hipGetLastError());
    check(hipMemcpy2D(hostPtr, hostPitch, devPtr, devPitch, hostPitch, n,
                       hipMemcpyDeviceToHost));
    sincelast("deviceToHost");
}

int main(int argc, char* argv[]) {
    MPI_Init(&argc, &argv);
    sincelast("placeholder");
    MPI_Comm_rank(MPI_COMM_WORLD, &R);
    int count;
    hipGetDeviceCount(&count);
    hipSetDevice(R < count ? R: count - 1);
    int B = atoi(argv[3]);
    input(argv[1]);
    mifn(B, XB);
    sincelast("input");
    blockfw(B);
    if (R == 0) {
        output(argv[2]);
        sincelast("output");
    }
    MPI_Finalize();
}
