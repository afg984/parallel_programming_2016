
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <fstream>

#define XB 24
const int INF = 10000000;

int n, m;  // Number of vertices, edges
size_t devPitch, hostPitch;
unsigned* hostPtr;

#define check(err) __check(err, __LINE__)
void __check(hipError_t err, int line) {
    if (err) {
        fprintf(stderr, "%d:%s\n", line, hipGetErrorString(err));
        abort();
    }
}

#define dist(i, j) (hostPtr[(i)*n + (j)])

void sincelast(const char* message = 0) {
    static auto last = std::chrono::high_resolution_clock::now();
    auto now = std::chrono::high_resolution_clock::now();
    fprintf(stderr, "%16s  %lf\n", message,
            ((std::chrono::duration<double>)(now - last)).count());
    last = now;
}

void input(char* inFileName) {
    FILE* infile = fopen(inFileName, "r");
    fscanf(infile, "%d %d", &n, &m);

    hostPitch = sizeof(int) * n;
    check(hipHostMalloc(&hostPtr, hostPitch * n, hipHostMallocDefault));

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j)
                dist(i, j) = 0;
            else
                dist(i, j) = INF;
        }
    }

    while (--m >= 0) {
        int a, b, v;
        fscanf(infile, "%u%u%u", &a, &b, &v);
        dist(a - 1, b - 1) = v;
    }
}

void output(char* outFileName) {
    std::ofstream fout(outFileName);
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (dist(i, j) >= INF)
                fout << "INF ";
            else
                fout << dist(i, j) << " ";
        }
        fout << "\n";
    }
}

#define safeat(i, j) ((i) < n and (j) < n ? at(i, j): INF)
#define at(i, j) (((int*)((char*)devPtr + (i)*devPitch))[j])
#define mifn(p, q)                \
    {                             \
        if ((p) > (q)) (p) = (q); \
    }

__global__ void phase1(unsigned* devPtr, int r, int n, int kmin, int devPitch) {
    int i = kmin + threadIdx.y;
    int j = kmin + threadIdx.x;
    if (i >= n or j >= n) {
        return;
    }
    int kmax = kmin + blockDim.x < n ? blockDim.x : n - kmin;
    __shared__ unsigned pivot[XB][XB];
    pivot[threadIdx.y][threadIdx.x] = at(i, j);
    for (int k = 0; k < kmax; ++k) {
        __syncthreads();
        mifn(pivot[threadIdx.y][threadIdx.x],
             pivot[threadIdx.y][k] + pivot[k][threadIdx.x]);
    }
    at(i, j) = pivot[threadIdx.y][threadIdx.x];
}

__global__ void phase2a(unsigned* devPtr, int r, int n, int kmin,
                        int devPitch) {
    if (blockIdx.y == r) {
        return;
    }
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = r * blockDim.x + threadIdx.x;
    int kmax = kmin + blockDim.x < n ? blockDim.x : n - kmin;
    __shared__ unsigned pivot[XB][XB], block[XB][XB];
    pivot[threadIdx.y][threadIdx.x] =
        safeat(kmin + threadIdx.y, kmin + threadIdx.x);
    block[threadIdx.y][threadIdx.x] = safeat(i, j);
    if (i >= n or j >= n) {
        return;
    }
    for (int k = 0; k < kmax; ++k) {
        __syncthreads();
        mifn(block[threadIdx.y][threadIdx.x],
             block[threadIdx.y][k] + pivot[k][threadIdx.x]);
    }
    at(i, j) = block[threadIdx.y][threadIdx.x];
}

__global__ void phase2b(unsigned* devPtr, int r, int n, int kmin,
                        int devPitch) {
    if (blockIdx.x == r) {
        return;
    }
    int i = r * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int kmax = kmin + blockDim.x < n ? blockDim.x : n - kmin;
    __shared__ unsigned pivot[XB][XB], block[XB][XB];
    pivot[threadIdx.y][threadIdx.x] =
        safeat(kmin + threadIdx.y, kmin + threadIdx.x);
    block[threadIdx.y][threadIdx.x] = safeat(i, j);
    if (i >= n or j >= n) {
        return;
    }
    for (int k = 0; k < kmax; ++k) {
        __syncthreads();
        mifn(block[threadIdx.y][threadIdx.x],
             block[k][threadIdx.x] + pivot[threadIdx.y][k]);
    }
    at(i, j) = block[threadIdx.y][threadIdx.x];
}

__global__ void phase3(unsigned* devPtr, int r, int n, int kmin, int devPitch) {
    if (blockIdx.x == r or blockIdx.y == r) {
        return;
    }
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int kmax = kmin + blockDim.x < n ? blockDim.x : n - kmin;
    __shared__ unsigned iref[XB][XB], jref[XB][XB];
    iref[threadIdx.y][threadIdx.x] = safeat(i, kmin + threadIdx.x);
    jref[threadIdx.y][threadIdx.x] = safeat(kmin + threadIdx.y, j);
    if (i >= n or j >= n) {
        return;
    }
    int local = at(i, j);
    __syncthreads();
    for (int k = 0; k < kmax; ++k) {
        mifn(local, iref[threadIdx.y][k] + jref[k][threadIdx.x]);
    }
    at(i, j) = local;
}

#undef at

int ceil(int a, int b) { return (a + b - 1) / b; }

void blockfw(int B) {
    unsigned* devPtr;
    check(hipMallocPitch(&devPtr, &devPitch, hostPitch, n));
    // printf("%d %d\n", devPitch, hostPitch);
    check(hipMemcpy2D(devPtr, devPitch, hostPtr, hostPitch, hostPitch, n,
                       hipMemcpyHostToDevice));
    sincelast("hostToDevice");
    int rounds = ceil(n, B);
    for (int r = 0; r < rounds; ++r) {
        int kmin = r * B;
        phase1<<<1, dim3(B, B)>>>(devPtr, r, n, kmin, devPitch);

        phase2a<<<dim3(1, rounds), dim3(B, B)>>>(devPtr, r, n, kmin, devPitch);
        phase2b<<<dim3(rounds, 1), dim3(B, B)>>>(devPtr, r, n, kmin, devPitch);

        phase3<<<dim3(rounds, rounds), dim3(B, B)>>>(devPtr, r, n, kmin,
                                                     devPitch);
    }
    check(hipDeviceSynchronize());
    sincelast("compute");
    check(hipGetLastError());
    check(hipMemcpy2D(hostPtr, hostPitch, devPtr, devPitch, hostPitch, n,
                       hipMemcpyDeviceToHost));
    sincelast("deviceToHost");
}

int main(int argc, char* argv[]) {
    sincelast("placeholder");
    int B = atoi(argv[3]);
    input(argv[1]);
    mifn(B, XB);
    sincelast("input");
    blockfw(B);
    output(argv[2]);
    sincelast("output");
}
